#include "parallel_huffman.h"
#include "utils.h"

// TODO: kernels for creating a histogram, sorting, mapping codes, and reducing (concatenating) codes

#define STREAM_SIZE 32768
#define STREAM_COUNT 3

void parallel_huffman_decode(
        const char* const input_buffer,
        unsigned int const& input_size,
        char* const output_buffer,
        unsigned int& output_size )
{
    // TODO: implement
}

// TODO: define static tree
static const unsigned char tree[] = {0};

void parallel_huffman_encode(
        const char* const input_buffer,
        unsigned int const& input_size,
        char* const output_buffer,
        unsigned int& output_size )
{
    hipStream_t streams[STREAM_COUNT];

    for ( int i = 0 ; i < STREAM_COUNT ; ++i ) {
        checkCudaErrors( hipStreamCreate( &streams[i] ) );
    }

    // TODO: allocate device memory for histogram buffer for sorting

    char* d_input;
    checkCudaErrors( hipMalloc( &d_input, input_size ) );

    char* d_output;
    checkCudaErrors( hipMalloc( &d_output, input_size ) );

    // TODO: update size and type for the tree

    unsigned int* d_output_size;
    checkCudaErrors( hipMalloc( &d_output_size, sizeof( *d_output_size ) ) );
    checkCudaErrors( hipMemsetAsync( d_output_size, 0, sizeof( *d_output_size ), streams[0] ) );

    char* d_tree;
    const unsigned int tree_size = sizeof( tree );
    checkCudaErrors( hipMalloc( &d_tree, tree_size ) );
    checkCudaErrors( hipMemcpyAsync( d_tree, tree, tree_size, hipMemcpyHostToDevice, streams[1] ) );

    char* d_histo;
    const unsigned int histo_size = 256 * sizeof( unsigned int );
    checkCudaErrors( hipMalloc( &d_histo, histo_size ) );
    checkCudaErrors( hipMemsetAsync( d_histo, 0, histo_size, streams[2] ) );

    // TODO: ensure there are no races from memset'ing and memcpy'ing

    // TODO: may want to do a dynamic number of streams?
    // TODO: may want to do a dynamic stream chunk size?

    checkCudaErrors( hipHostRegister( static_cast<void*>( const_cast<char*>( input_buffer ) ), input_size, hipHostRegisterMapped ) );

    unsigned int j = 0;

    for ( unsigned int i = 0 ; i < input_size ; i += STREAM_SIZE ) {
        const unsigned int chunk_size = ( i + STREAM_SIZE > input_size ? input_size - i : STREAM_SIZE );
        checkCudaErrors( hipMemcpyAsync( &d_input[i], &input_buffer[i], chunk_size, hipMemcpyHostToDevice, streams[j] ) );
        j = ( j + 1 ) % STREAM_COUNT;
    }

    // TODO: histogram kernel

    checkCudaErrors( hipDeviceSynchronize() );

    checkCudaErrors( hipHostUnregister( static_cast<void*>( const_cast<char*>( input_buffer ) ) ) );

    // TODO: sort

    // TODO: move huffman tree elements with sorted values

    // TODO: map input symbols to output symbols and compact

    // TODO: output the huffman table along with output data

    // TODO: update reduce input size to actual compressed size

#if 0 
    checkCudaErrors( hipMemcpy( &output_size, d_output_size, sizeof( *d_output_size ), hipMemcpyDeviceToHost ) );
#else
    output_size = input_size;
#endif

    j = 0;

    // TODO: update d_input with whatever buffer ends up being used

    checkCudaErrors( hipHostRegister( static_cast<void*>( output_buffer ), output_size, hipHostRegisterMapped ) );

    for ( unsigned int i = 0 ; i < output_size ; i += STREAM_SIZE ) {
        const unsigned int chunk_size = ( i + STREAM_SIZE > output_size ? output_size - i : STREAM_SIZE );
        checkCudaErrors( hipMemcpyAsync( &output_buffer[i], &d_input[i], chunk_size, hipMemcpyDeviceToHost, streams[j] ) );
        j = ( j + 1 ) % STREAM_COUNT;
    }

    checkCudaErrors( hipDeviceSynchronize() );

    checkCudaErrors( hipHostUnregister( static_cast<void*>( output_buffer ) ) );

    // TODO: release any additional memory allocated

    checkCudaErrors( hipFree( d_input ) );
    checkCudaErrors( hipFree( d_output ) );
    checkCudaErrors( hipFree( d_output_size ) );
    checkCudaErrors( hipFree( d_tree ) );
    checkCudaErrors( hipFree( d_histo ) );

    for ( int i = 0 ; i < STREAM_COUNT ; ++i ) {
        checkCudaErrors( hipStreamDestroy( streams[i] ) );
    }
}
