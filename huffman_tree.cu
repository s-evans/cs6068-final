#include "hip/hip_runtime.h"
#include "huffman_tree.h"
#include "blelloch_scan.h"
#include "utils.h"

#include <ostream>

__device__ unsigned int d_start_idx;

__global__ void find_start_idx(
        const unsigned int* const sorted_histogram )
{
    const unsigned int idx = threadIdx.x;

    if ( sorted_histogram[idx] == 0 ) {
        atomicAdd( &d_start_idx, 1 );
    }
}

__global__ void initialize_nodes(
        const unsigned int* const sorted_histogram,
        const unsigned int* const sorted_symbols,
        node_t* const nodes,
        const unsigned int populated_nodes
        )
{
    const unsigned int idx = threadIdx.x;

    if ( idx < populated_nodes ) {
        nodes[idx].left_idx = USHRT_MAX;
        nodes[idx].symbol = static_cast<short>( sorted_symbols[idx] );
        nodes[idx].weight = sorted_histogram[idx];
    } else {
        nodes[idx].left_idx = USHRT_MAX;
        nodes[idx].symbol = 0;
        nodes[idx].weight = 0;
    }

}

std::ostream& operator<< ( std::ostream& stream, node_t const& node )
{
    return stream << "node.left_idx: " << static_cast<unsigned int>( node.left_idx ) 
        << "; node.symbol: "  << static_cast<unsigned int>( node.symbol ) 
        << "; node.weight: " << static_cast<unsigned int>( node.weight ) 
        << ";";
}

__global__ void insert_super_node (
        node_t* const nodes,
        unsigned int offset )
{
    __shared__ unsigned int not_moved;
    __shared__ unsigned int left_weight;
    __shared__ unsigned int right_weight;
    __shared__ unsigned int new_weight;
    const unsigned int idx = threadIdx.x;

    if ( idx == 0 ) {
        not_moved = 0;
        left_weight = nodes[0].weight;
        right_weight = nodes[1].weight;
        new_weight = left_weight + right_weight;
    }

    __syncthreads();

    if ( left_weight == 0 || right_weight == 0 ) {
        __syncthreads();
        return;
    }

    const unsigned int my_weight = nodes[idx].weight;

    if ( my_weight == 0 ) {
        __syncthreads();
        return;
    }

    const bool move = ( my_weight >= new_weight );
    node_t node;

    if ( move ) {
        node = nodes[idx];
    } else {
        atomicAdd( &not_moved, 1 );
    }

    __syncthreads();

    if ( move ) {
        nodes[idx + 1] = node;
    } else if ( idx == 0 ) {
        node_t* const pnode = &nodes[not_moved];
        pnode->left_idx = offset;
        pnode->symbol = 0;
        pnode->weight = new_weight;
    }
}

void make_huffman_tree(
        const unsigned int* const d_sorted_histogram,
        const unsigned int* const d_sorted_symbols,
        const unsigned int histogram_size)
{
    // TODO: free device buffer eventually
    // TODO: do something useful with the tree

    void* p_d_start_idx; 
    checkCudaErrors( hipGetSymbolAddress( &p_d_start_idx, HIP_SYMBOL(d_start_idx )) );
    checkCudaErrors( hipMemsetAsync( p_d_start_idx, 0, sizeof( d_start_idx ), 0 ) );

    find_start_idx<<<1, histogram_size, 0, 0>>>( d_sorted_histogram );

    unsigned int start_idx;
    checkCudaErrors( hipMemcpyAsync( &start_idx, p_d_start_idx, sizeof( start_idx ), hipMemcpyDeviceToHost, 0 ) );

    checkCudaErrors( hipStreamSynchronize( 0 ) );

    const unsigned int node_count = histogram_size - start_idx;
    const unsigned int max_node_count = ( blelloch_size( node_count ) << 1 ) - 1;

    node_t* d_nodes;
    checkCudaErrors( hipMalloc( &d_nodes, sizeof( *d_nodes ) * max_node_count ) );
    initialize_nodes<<<1, max_node_count, 0, 0>>>(
            &d_sorted_histogram[start_idx], 
            &d_sorted_symbols[start_idx],
            d_nodes,
            node_count );

    for ( unsigned int i = 0 ; i < max_node_count ; i += 2 ) {
        insert_super_node<<<1, max_node_count - i, 0, 0>>>( &d_nodes[i], i );
    }

    // TODO: get idx of head (ie. furthest right node) (ie. highest weight node)

    /* std::cerr << "node_count: " << node_count << std::endl; */
    /* std::cerr << "max_node_count: " << max_node_count << std::endl; */
    /* checkCudaErrors( hipStreamSynchronize( 0 ) ); */

    /* for ( unsigned int i = 0 ; i < max_node_count ; ++i ) { */
    /*     node_t tmp_node; */
    /*     checkCudaErrors( hipMemcpy( &tmp_node, &d_nodes[i], sizeof( tmp_node ), hipMemcpyDeviceToHost ) ); */
    /*     std::cerr << "idx: " << i << "; " << tmp_node << std::endl; */
    /* } */
}
